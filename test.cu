#include "hip/hip_runtime.h"


#include "cute/algorithm/copy.hpp"

#include "cutlass/cutlass.h"
#include "cutlass/layout/layout.h"
#include <cutlass/numeric_types.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

// #include <torch/torch.h>
// #include <ATen/ATen.h>
// #include <ATen/cuda/HIPContext.h>
// #include <ATen/cuda/detail/CUDAHooks.h>
// #include <torch/nn/functional.h>
// #include <c10/cuda/CUDAGuard.h>


#include <cute/tensor.hpp> 

using namespace cute;


constexpr int TOTAL_DIM = 0;
constexpr int H_DIM = 1;
constexpr int D_DIM = 2;

////////////////////////////////////////////////////////////////////////////////////////////////////

struct Qkv_params {
    using index_t = int64_t;
    // The QKV matrices.
    void *__restrict__ q_ptr;
    void *__restrict__ k_ptr;
    void *__restrict__ v_ptr;

    // The stride between rows of the Q, K and V matrices.
    index_t q_batch_stride;
    index_t k_batch_stride;
    index_t v_batch_stride;
    index_t q_row_stride;
    index_t k_row_stride;
    index_t v_row_stride;
    index_t q_head_stride;
    index_t k_head_stride;
    index_t v_head_stride;

    // The number of heads.
    int h, h_k;
    // In the case of multi-query and grouped-query attention (MQA/GQA), nheads_k could be
    // different from nheads (query).
    int h_h_k_ratio; // precompute h / h_k,
};

////////////////////////////////////////////////////////////////////////////////////////////////////

struct Flash_fwd_params : public Qkv_params {

    // The O matrix (output).
    void * __restrict__ o_ptr;
    void * __restrict__ oaccum_ptr;

    // The stride between rows of O.
    index_t o_batch_stride;
    index_t o_row_stride;
    index_t o_head_stride;

    // The pointer to the P matrix.
    void * __restrict__ p_ptr;

    // The pointer to the softmax sum.
    void * __restrict__ softmax_lse_ptr;
    void * __restrict__ softmax_lseaccum_ptr;

    // The dimensions.
    int b, seqlen_q, seqlen_k, seqlen_knew, d, seqlen_q_rounded, seqlen_k_rounded, d_rounded, rotary_dim;

    // The scaling factors for the kernel.
    float scale_softmax;
    float scale_softmax_log2;

    // array of length b+1 holding starting offset of each sequence.
    int * __restrict__ cu_seqlens_q;
    int * __restrict__ cu_seqlens_k;

    // If provided, the actual length of each k sequence.
    int * __restrict__ seqused_k;

    int *__restrict__ blockmask;

    // The K_new and V_new matrices.
    void * __restrict__ knew_ptr;
    void * __restrict__ vnew_ptr;

    // The stride between rows of the Q, K and V matrices.
    index_t knew_batch_stride;
    index_t vnew_batch_stride;
    index_t knew_row_stride;
    index_t vnew_row_stride;
    index_t knew_head_stride;
    index_t vnew_head_stride;

    // The cos and sin matrices for rotary embedding.
    void * __restrict__ rotary_cos_ptr;
    void * __restrict__ rotary_sin_ptr;

    // The indices to index into the KV cache.
    int * __restrict__ cache_batch_idx;

    // Paged KV cache
    int * __restrict__ block_table;
    index_t block_table_batch_stride;
    int page_block_size;

    // The dropout probability (probability of keeping an activation).
    float p_dropout;
    // uint32_t p_dropout_in_uint;
    // uint16_t p_dropout_in_uint16_t;
    uint8_t p_dropout_in_uint8_t;

    // Scale factor of 1 / (1 - p_dropout).
    float rp_dropout;
    float scale_softmax_rp_dropout;

    // Local window size
    int window_size_left, window_size_right;

    // // Random state.
    // at::PhiloxCudaState philox_args;

    // Pointer to the RNG seed (idx 0) and offset (idx 1).
    uint64_t * rng_state;

    bool is_bf16;
    bool is_causal;

    // If is_seqlens_k_cumulative, then seqlen_k is cu_seqlens_k[bidb + 1] - cu_seqlens_k[bidb].
    // Otherwise it's cu_seqlens_k[bidb], i.e., we use cu_seqlens_k to store the sequence lengths of K.
    bool is_seqlens_k_cumulative;

    bool is_rotary_interleaved;

    int num_splits;  // For split-KV version

    void * __restrict__ alibi_slopes_ptr;
    index_t alibi_slopes_batch_stride;
};




template<int kHeadDim_, int kBlockM_, int kBlockN_, int kNWarps_, typename elem_type=cutlass::half_t>
struct Flash_kernel_traits {

#if defined(__CUDA_ARCH__) &&  __CUDA_ARCH__ >= 800
    using Element = elem_type;
    static constexpr bool Has_cp_async = true;
#else
    using Element = cutlass::half_t;
    static constexpr bool Has_cp_async = false;
#endif

    using ElementAccum = float;
    using index_t = int64_t;

#if defined(__CUDA_ARCH__) &&  __CUDA_ARCH__ >= 800
#pragma message("cuda arch >= 800")
    using MMA_Atom_Arch = std::conditional_t<
        std::is_same_v<elem_type, cutlass::half_t>,
        MMA_Atom<SM80_16x8x16_F32F16F16F32_TN>,
        MMA_Atom<SM80_16x8x16_F32BF16BF16F32_TN>
    >;
// #else 
//     using MMA_Atom_Arch = MMA_Atom<SM75_16x8x8_F32F16F16F32_TN>;
#elif defined(__CUDA_ARCH__) &&  __CUDA_ARCH__ == 750
#pragma message("cuda arch == 750")
    using MMA_Atom_Arch = MMA_Atom<SM75_16x8x8_F32F16F16F32_TN>;
#elif defined(__CUDA_ARCH__) &&  __CUDA_ARCH__ == 700
#pragma message("cuda arch == 700")
    using MMA_Atom_Arch = MMA_Atom<SM70_8x8x4_F32F16F16F32_TN>;
#else
#pragma message("cuda arch < 700")
    using MMA_Atom_Arch = MMA_Atom<SM61_DP4A>;
#endif

#if defined(__CUDA_ARCH__) &&  __CUDA_ARCH__ >= 750
    using SmemCopyAtom = Copy_Atom<SM75_U32x4_LDSM_N, elem_type>;
    using SmemCopyAtomTransposed = Copy_Atom<SM75_U16x8_LDSM_T, elem_type>;
#else
    using SmemCopyAtom = Copy_Atom<DefaultCopy, elem_type>;
    using SmemCopyAtomTransposed = Copy_Atom<DefaultCopy, elem_type>;
#endif
};

// If Share_Q_K_smem is true, that forces Is_Q_in_regs to be true
template<int kHeadDim_, int kBlockM_, int kBlockN_, int kNWarps_, bool Is_Q_in_regs_=false, bool Share_Q_K_smem_=false, typename elem_type=cutlass::half_t,
         typename Base=Flash_kernel_traits<kHeadDim_, kBlockM_, kBlockN_, kNWarps_, elem_type> >
struct Flash_fwd_kernel_traits : public Base {
    using Element = typename Base::Element;
    using ElementAccum = typename Base::ElementAccum;
    using index_t = typename Base::index_t;
    static constexpr bool Has_cp_async = Base::Has_cp_async;
    using SmemCopyAtom = typename Base::SmemCopyAtom;
    using SmemCopyAtomTransposed = typename Base::SmemCopyAtomTransposed;

    static constexpr bool Share_Q_K_smem = Share_Q_K_smem_;
    static constexpr bool Is_Q_in_regs = Is_Q_in_regs_ || Share_Q_K_smem;

    // The number of threads.
    static constexpr int kNWarps = kNWarps_;   // 4
    static constexpr int kNThreads = kNWarps * 32;  // 4*32 = 128

    static constexpr int kBlockM = kBlockM_;  // 128
    static constexpr int kBlockN = kBlockN_;  // 64
    static constexpr int kHeadDim = kHeadDim_;  // 64
    static_assert(kHeadDim % 32 == 0);
    static constexpr int kBlockKSmem = kHeadDim % 64 == 0 ? 64 : 32;  // 64
    static constexpr int kBlockKGmem = kHeadDim % 128 == 0 ? 128 : (kHeadDim % 64 == 0 ? 64 : 32); // 64
    static constexpr int kSwizzle = kBlockKSmem == 32 ? 2 : 3;  // 3

    // 16x8x16, 4x1x1, 64x16x16
    using TiledMma = TiledMMA<
        typename Base::MMA_Atom_Arch,
        Layout<Shape<Int<kNWarps>,_1,_1>>,  // 4x1x1 or 8x1x1 thread group
        Tile<Int<16 * kNWarps>, _16, _16>>;

    // <3x3x3> <8,64> <64,1>
    using SmemLayoutAtomQ = decltype(
        composition(Swizzle<kSwizzle, 3, 3>{},
                    // This has to be kBlockKSmem, using kHeadDim gives wrong results for d=128
                    Layout<Shape<_8, Int<kBlockKSmem>>,
                           Stride<Int<kBlockKSmem>, _1>>{}));
    using SmemLayoutQ = decltype(tile_to_shape(
        SmemLayoutAtomQ{},
        Shape<Int<kBlockM>, Int<kHeadDim>>{}));

    using SmemLayoutKV = decltype(tile_to_shape(
        SmemLayoutAtomQ{},
        Shape<Int<kBlockN>, Int<kHeadDim>>{}));

    // https://github.com/ColfaxResearch/cutlass-kernels/blob/a222587e6d59b93ba704853d3946fb686d8b8892/src/fmha/fmha_forward.cu#L434
    using SmemLayoutVtransposed = decltype(
        composition(SmemLayoutKV{}, make_layout(Shape<Int<kHeadDim>, Int<kBlockN>>{}, GenRowMajor{})));
    using SmemLayoutVtransposedNoSwizzle = decltype(get_nonswizzle_portion(SmemLayoutVtransposed{}));

    using SmemLayoutAtomO = decltype(
        composition(Swizzle<kSwizzle, 3, 3>{},
                    Layout<Shape<Int<8>, Int<kBlockKSmem>>,
                           Stride<Int<kBlockKSmem>, _1>>{}));
    using SmemLayoutO = decltype(tile_to_shape(
        SmemLayoutAtomO{},
        Shape<Int<kBlockM>, Int<kHeadDim>>{}));
    using SmemCopyAtomO = Copy_Atom<DefaultCopy, Element>;
    using SmemCopyAtomOaccum = Copy_Atom<DefaultCopy, ElementAccum>;

    static constexpr int kSmemQSize = size(SmemLayoutQ{}) * sizeof(Element);
    static constexpr int kSmemKVSize = size(SmemLayoutKV{}) * 2 * sizeof(Element);
    static constexpr int kSmemSize = Share_Q_K_smem ? std::max(kSmemQSize, kSmemKVSize) : kSmemQSize + kSmemKVSize;

    static constexpr int kGmemElemsPerLoad = sizeof(cute::uint128_t) / sizeof(Element);
    static_assert(kHeadDim % kGmemElemsPerLoad == 0, "kHeadDim must be a multiple of kGmemElemsPerLoad");
    // Using kBlockKSmem here is 6-10% faster than kBlockKGmem for d=128 because of bank conflicts.
    // For example, for d=128, smem is split into 2 "pages", each page takes care of columns
    // 0-63 and 64-127. If we have 16 threads per row for gmem read, when we write to smem,
    // thread 0 - 7 will write to the first page and thread 8 - 15 will write to the second page,
    // to the same banks.
    static constexpr int kGmemThreadsPerRow = kBlockKSmem / kGmemElemsPerLoad;
    static_assert(kNThreads % kGmemThreadsPerRow == 0, "kNThreads must be a multiple of kGmemThreadsPerRow");
    using GmemLayoutAtom = Layout<Shape <Int<kNThreads / kGmemThreadsPerRow>, Int<kGmemThreadsPerRow>>,
                                  Stride<Int<kGmemThreadsPerRow>, _1>>;

    // We use CACHEGLOBAL instead of CACHEALWAYS for both Q and K/V, since we won't be reading
    // from the same address by the same threadblock. This is slightly faster.
    using Gmem_copy_struct = std::conditional_t<
        Has_cp_async,
        SM80_CP_ASYNC_CACHEGLOBAL<cute::uint128_t>,
        DefaultCopy
    >;
    using GmemTiledCopyQKV = decltype(
        make_tiled_copy(Copy_Atom<Gmem_copy_struct, Element>{},
                        GmemLayoutAtom{},
                        Layout<Shape<_1, _8>>{}));  // Val layout, 8 vals per read
    using GmemTiledCopyO = decltype(
        make_tiled_copy(Copy_Atom<DefaultCopy, Element>{},
                        GmemLayoutAtom{},
                        Layout<Shape<_1, _8>>{}));  // Val layout, 8 vals per store

    using GmemLayoutAtomOaccum = std::conditional_t<
        kBlockKSmem == 32,
        Layout<Shape <_16, _8>,  // Thread layout, 8 threads per row
               Stride< _8, _1>>,
        Layout<Shape <_8, _16>,  // Thread layout, 16 threads per row
               Stride< _16, _1>>
    >;
    using GmemTiledCopyOaccum = decltype(
        make_tiled_copy(Copy_Atom<DefaultCopy, ElementAccum>{},
                        GmemLayoutAtomOaccum{},
                        Layout<Shape < _1, _4>>{}));  // Val layout, 4 vals per store
    using GmemLayoutAtomRotcossin = GmemLayoutAtom;
    using GmemTiledCopyRotcossin = decltype(
        make_tiled_copy(Copy_Atom<UniversalCopy<uint64_t>, Element>{},
                        GmemLayoutAtomRotcossin{},
                        Layout<Shape < _1, _4>>{}));  // Val layout, 4 vals per load
    using GmemTiledCopyRotcossinCont = decltype(
        make_tiled_copy(Copy_Atom<DefaultCopy, Element>{},
                        GmemLayoutAtomRotcossin{},
                        Layout<Shape < _1, _8>>{}));  // Val layout, 8 vals per load
};



/// Simple copy kernel.
//
// Uses local_partition() to partition a tile among threads arranged as (THR_M, THR_N).
template <class TensorS, class TensorD, class ThreadLayout>
__global__ void copy_kernel(TensorS S, TensorD D, ThreadLayout) 
{
  using namespace cute;

  // Slice the tiled tensors
  Tensor tile_S = S(make_coord(_,_), blockIdx.x, blockIdx.y);   // (BlockShape_M, BlockShape_N)
  Tensor tile_D = D(make_coord(_,_), blockIdx.x, blockIdx.y);   // (BlockShape_M, BlockShape_N)

  // Construct a partitioning of the tile among threads with the given thread arrangement.

  // Concept:                       Tensor    Layout          Index
  Tensor thr_tile_S = local_partition(tile_S, ThreadLayout{}, threadIdx.x);
  Tensor thr_tile_D = local_partition(tile_D, ThreadLayout{}, threadIdx.x);

  // Construct a register-backed Tensor with the same shape as each thread's partition
  auto fragment = make_fragment_like(thr_tile_S);

  // Copy from GMEM to RMEM and from RMEM to GMEM
  copy(thr_tile_S, fragment);
  copy(fragment, thr_tile_D);
}

/// Vectorized copy kernel.
///
/// Uses `make_tiled_copy()` to perform a copy using vector instructions. This operation
/// has the precondition that pointers are aligned to the vector size.
///
template <class TensorS, class TensorD, class ThreadLayout, class VecLayout>
__global__ void copy_kernel_vectorized(TensorS S, TensorD D, ThreadLayout, VecLayout) 
{
  using namespace cute;
  using Element = typename TensorS::value_type;

  // Slice the tensors to obtain a view into each tile.
  Tensor tile_S = S(make_coord(_, _), blockIdx.x, blockIdx.y);   // (BlockShape_M, BlockShape_N)
  Tensor tile_D = D(make_coord(_, _), blockIdx.x, blockIdx.y);   // (BlockShape_M, BlockShape_N)

  // Define `AccessType` which controls the size of the actual memory access.
  using AccessType = cutlass::AlignedArray<Element, size(shape(VecLayout{}))>;

  // A copy atom corresponds to one hardware memory access.
  using Atom = Copy_Atom<UniversalCopy<AccessType>, Element>;

  // Construct tiled copy, a tiling of copy atoms.
  //
  // Note, this assumes the vector and thread layouts are aligned with contigous data
  // in GMEM. Alternative thread layouts are possible but may result in uncoalesced
  // reads. Alternative vector layouts are also possible, though incompatible layouts
  // will result in compile time errors.
  auto tiled_copy =
    make_tiled_copy(
      Atom{},                       // access size
      ThreadLayout{},               // thread layout
      VecLayout{});                 // vector layout (e.g. 4x1)

  // Construct a Tensor corresponding to each thread's slice.
  auto thr_copy = tiled_copy.get_thread_slice(threadIdx.x);

  Tensor thr_tile_S = thr_copy.partition_S(tile_S);
  Tensor thr_tile_D = thr_copy.partition_D(tile_D);

  // Construct a register-backed Tensor with the same shape as each thread's partition
  auto fragment = make_fragment_like(thr_tile_D);

  // Copy from GMEM to RMEM and from RMEM to GMEM
  copy(tiled_copy, thr_tile_S, fragment);
  copy(tiled_copy, fragment, thr_tile_D);
}

/// Helper to convert a shape to a dim3
template <class Shape>
dim3 shape_to_dim3(Shape shape)
{
  using namespace cute;

  CUTE_STATIC_ASSERT_V(rank(shape) <= Int<3>{});
  auto result = append<3>(product_each(shape), 1u);

  return dim3(get<0>(result), get<1>(result), get<2>(result));
}



template<bool Varlen=true, int seqlen_q=1024, int seqlen_k=1024>
struct BlockInfo {

    __device__ BlockInfo(const int bidb)
        : sum_s_q(-1)
        , sum_s_k(-1)
        , actual_seqlen_q(seqlen_q)
        // If is_seqlens_k_cumulative, then seqlen_k is cu_seqlens_k[bidb + 1] - cu_seqlens_k[bidb].
        // Otherwise it's cu_seqlens_k[bidb], i.e., we use cu_seqlens_k to store the sequence lengths of K.
        , seqlen_k_cache(seqlen_k)
        , actual_seqlen_k(seqlen_k_cache)
        {
        }

    template <typename index_t>
    __forceinline__ __device__ index_t q_offset(const index_t batch_stride, const index_t row_stride, const int bidb) const {
        return sum_s_q == -1 ? bidb * batch_stride : uint32_t(sum_s_q) * row_stride;
    }

    template <typename index_t>
    __forceinline__ __device__ index_t k_offset(const index_t batch_stride, const index_t row_stride, const int bidb) const {
        return sum_s_k == -1 ? bidb * batch_stride : uint32_t(sum_s_k) * row_stride;
    }

    const int sum_s_q;
    const int sum_s_k;
    const int actual_seqlen_q;
    // We have to have seqlen_k_cache declared before actual_seqlen_k, otherwise actual_seqlen_k is set to 0.
    const int seqlen_k_cache;
    const int actual_seqlen_k;
};



template<typename Kernel_traits>
inline __device__ void compute_attn_1rowblock(float* Q, float* K, float* V, float* O, const int bidb, const int bidh, const int m_block) {

    using Element = typename Kernel_traits::Element;
    using ElementAccum = typename Kernel_traits::ElementAccum;
    using index_t = typename Kernel_traits::index_t;

    // Shared memory.
    extern __shared__ char smem_[];

    // The thread index.
    const int tidx = threadIdx.x;

    constexpr int kBlockM = Kernel_traits::kBlockM;//128
    constexpr int kBlockN = Kernel_traits::kBlockN;//64
    constexpr int kHeadDim = Kernel_traits::kHeadDim;//64
    constexpr int kNWarps = Kernel_traits::kNWarps;//4

    // params.q_ptr = q.data_ptr();
    // params.k_ptr = k.data_ptr();
    // params.v_ptr = v.data_ptr();
    // All stride are in elements, not bytes.
    //2097152, 2048, 64, 1
    int seqlen_q = 1024;
    int q_row_stride = 1;
    int k_row_stride = 1;
    int v_row_stride = 1;
    int q_head_stride = 64;
    int k_head_stride = 64;
    int v_head_stride = 64;
    // params.o_ptr = out.data_ptr();
    int o_row_stride = 1;
    int o_head_stride = 64;

        int q_batch_stride = 2097152;
        int k_batch_stride = 2097152;
        int v_batch_stride = 2097152;
        int o_batch_stride = 2097152;
        

    const BlockInfo</*Varlen=*/false> binfo(bidb);
    if (m_block * kBlockM >= binfo.actual_seqlen_q) return;

        const index_t row_offset_o = binfo.q_offset(o_batch_stride, o_row_stride, bidb)
            + m_block * kBlockM * o_row_stride + bidh * o_head_stride;
        const index_t row_offset_lse = (bidb * kHeadDim + bidh) * seqlen_q + m_block * kBlockM;
        Tensor gO = make_tensor(make_gmem_ptr(reinterpret_cast<Element *>(O) + row_offset_o),
                                Shape<Int<kBlockM>, Int<kHeadDim>>{},
                                make_stride(o_row_stride, _1{}));

        typename Kernel_traits::GmemTiledCopyO gmem_tiled_copy_O;
        auto gmem_thr_copy_O = gmem_tiled_copy_O.get_thread_slice(tidx);
        Tensor tOgO = gmem_thr_copy_O.partition_D(gO);
        Tensor tOrO = make_tensor<Element>(shape(tOgO));
        clear(tOrO);
        // Construct identity layout for sO
        Tensor cO = make_identity_tensor(make_shape(size<0>(gO), size<1>(gO)));    // (BLK_M,BLK_K) -> (blk_m,blk_k)
        // Repeat the partitioning with identity layouts
        Tensor tOcO = gmem_thr_copy_O.partition_D(cO);
        Tensor tOpO = make_tensor<bool>(make_shape(size<2>(tOgO)));

        typename Kernel_traits::TiledMma tiled_mma;
        auto thr_mma = tiled_mma.get_thread_slice(tidx);
        Tensor acc_o = partition_fragment_C(tiled_mma, Shape<Int<kBlockM>, Int<kHeadDim>>{});  // MMA, MMA_M, MMA_K

}

template<typename Kernel_traits>
inline __device__ void compute_attn(float* Q, float* K, float* V, float* O) {
    const int m_block = blockIdx.x;
    // The block index for the batch.
    const int bidb = blockIdx.y;
    // The block index for the head.
    const int bidh = blockIdx.z;

    // We want the fwd and bwd to generate the same dropout pattern (RNG), without restricting
    // them to have the same number of threads or have to traverse the attention matrix
    // in the same order.
    // In the Philox RNG, we use the offset to store the batch, head, and the lane id
    // (within a warp). We use the subsequence to store the location of the 16 x 32 blocks within
    // the attention matrix. This way, as long as we have the batch, head, and the location of
    // the 16 x 32 block within the attention matrix, we can generate the exact same dropout pattern.

    compute_attn_1rowblock<Kernel_traits>(Q, K, V, O, bidb, bidh, m_block);
}

template<typename Kernel_traits>
__global__ void flash_fwd_kernel(float* Q, float* K, float* V, float* O) {
    compute_attn<Kernel_traits>(Q, K, V, O);
}



int main() {


  using namespace cute;
  using Element = float;

    const int batch_size = 2;
    const int seq_len = 1024;
    const int num_heads = 32;
    const int head_dim = 64;
    const int seqlen_q = seq_len;
    const int seqlen_k = seq_len;
    const int seqlen_q_rounded = seqlen_q;
    const int seqlen_k_rounded = seqlen_k;
    const int num_heads_k = head_dim;
    const int head_size = head_dim;
    const int head_size_rounded = head_size;

  auto tensor_shape1 = make_shape(batch_size, seqlen_q, num_heads, head_dim);
  thrust::host_vector<Element> h_Q(size(tensor_shape1));
  thrust::host_vector<Element> h_K(size(tensor_shape1));
  thrust::host_vector<Element> h_V(size(tensor_shape1));
  thrust::host_vector<Element> h_O(size(tensor_shape1));

  for (size_t i = 0; i < h_Q.size(); ++i) {
    h_Q[i] = static_cast<Element>(i);
    h_K[i] = static_cast<Element>(i);
    h_V[i] = static_cast<Element>(i);
    h_O[i] = static_cast<Element>(i);
  }

  thrust::device_vector<Element> d_Q = h_Q;
  thrust::device_vector<Element> d_K = h_K;
  thrust::device_vector<Element> d_V = h_V;
  thrust::device_vector<Element> d_O = h_O;

    using Kernel_traits = Flash_fwd_kernel_traits<32, 128, 64, 4>;
    // typename Kernel_traits::TiledMma tiled_mma;
    // int tidx = 10;
    // auto thr_mma = tiled_mma.get_thread_slice(tidx);

    constexpr size_t smem_size = Kernel_traits::kSmemSize;
    printf("smem_size = %d\n", smem_size);

    const int num_m_block = (seq_len + Kernel_traits::kBlockM - 1) / Kernel_traits::kBlockM;
    dim3 grid(num_m_block, batch_size, num_heads);
    printf("%d, %d, %d\n", num_m_block, batch_size, num_heads);

    auto kernel = &flash_fwd_kernel<Kernel_traits>;
    kernel<<<grid, Kernel_traits::kNThreads, smem_size>>>(d_Q.data().get(), d_K.data().get(), d_V.data().get(), d_O.data().get());



    return 0;
}























